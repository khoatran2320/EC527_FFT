/*****************************************************************************/
// nvcc -arch sm_35 mmm_shared.cu -o mmm_shared


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <complex.h>
#include <stdint.h>
#include <string.h>
#include <stdbool.h>
#include <stddef.h>
#include <assert.h>
#include <hip/hip_complex.h>

#define GIG 1000000000
#define CPG 3.0           // Cycles per GHz -- Adjust to your computer

#define TILE_WIDTH 32
#define SIZE 8192
#define BLOCK_SIZE (SIZE / 8)
#define NUM_BLOCKS 8
#define SAMPLING_RATE 100
#define FREQUENCY 2

#define OPTIONS 1
#define IDENT 0


static void generate_sin_points_c(hipDoubleComplex *v, unsigned long size, unsigned long sampling_rate, unsigned long frequency){
    double time = 0.0;
    double inc = (double)1/sampling_rate;
    double W = (double)2 * M_PI * frequency;
    int i;

    for(i = 0; i < size; ++i){
        v[i] = make_hipDoubleComplex(100*sin(time*W), 0);
		//printf("v[%d] is %.2lf j%.2lf\n", i, creal(v[i]), cimag(v[i])); //Used for debugging, checked why vector results in function didn't match result outside function
        time += inc;
    }
}

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

//My CUDA function for global FFT that works on entire matrix
__global__ void kernel_FFT (int rowlen, hipDoubleComplex * exptable, hipDoubleComplex * fft_matrix) {
  
    int levels = 0;
    hipDoubleComplex temp;
    int i, j, k, l, m, size;
    int val;

	for (i = rowlen; i > 1U; i >>= 1)
		levels++;
	
	// Bit-reversed addressing permutation
	// 000 => 000
	// 001 => 100
	// 010 => 010
	// 011 => 110
	// ...
	// for n=8: [a0, a1, a2, a3, a4, a5, a6, a7] => [a0, a4, a2, a6, a1, a5, a3, a7]
	int i_index;
	int j_index;
	int l_index;

    /* Determine the row number thread is acting on and precompute stride accordingly */
    int row = blockIdx.x * (rowlen / gridDim.x) + threadIdx.x;
    int stride = row * rowlen;


    /* Swap the vector elements */
    for (i = 0; i < rowlen; i++) {
        val = i;
        j = 0;
        for (k = 0; k < levels; k++, val >>= 1)
            j = (j << 1) | (val & 1U);

        if (j > i) {
            i_index = stride + i;
            j_index = stride + j;
            //printf("Row %d i_index = %d, j_index = %d\n", row, i_index, j_index);
            //printf("Row %d fft_matrix[i(%d)] = %.2lf j%.2lf & fft_matrix[j(%d)] = %.2lf j%.2lf\n", row, i, cuCreal(fft_matrix[i_index]), cuCimag(fft_matrix[i_index]), j, cuCreal(fft_matrix[j_index]), cuCimag(fft_matrix[j_index]));
            hipDoubleComplex temp = fft_matrix[i_index];
            fft_matrix[i_index] = fft_matrix[j_index];
            fft_matrix[j_index] = temp;
            //printf("Afterwards: Row %d fft_matrix[i(%d)] = %.2lf j%.2lf & fft_matrix[j(%d)] = %.2lf j%.2lf\n", row, i, cuCreal(fft_matrix[i_index]), cuCimag(fft_matrix[i_index]), j, cuCreal(fft_matrix[j_index]), cuCimag(fft_matrix[j_index]));
        }
    }
    
    // Cooley-Tukey decimation-in-time radix-2 FFT
    // loop through each stage

    for (size = 2; size <= rowlen; size *= 2) {														
        int halfsize = size / 2;																			
        int tablestep = rowlen / size;		

        //for each stage, compute butterly for 2 outputs in groups of 2, 4, 8, ...															
        for (i = 0; i < rowlen; i += size) {	
            // compute butterfly (2 outputs)														

            for (j = i, k = 0; j < i + halfsize; j++, k += tablestep) {
                int l = j + halfsize;
                j_index = (stride) + j;
                l_index = (stride) + l;									
                temp = hipCmul( fft_matrix[l_index] , exptable[k]);
                //printf("Multiplication result is: %.2lf j%.2lf ", cuCreal(temp), cuCimag(temp));
                fft_matrix[l_index] = hipCsub(fft_matrix[j_index], temp);
                fft_matrix[j_index] = hipCadd(fft_matrix[j_index], temp);
            }

            if (size == rowlen)  // Prevent overflow in 'size *= 2'
                break;
        }
    }
}







//My CUDA function for matrix transpose (in place)
__global__ void kernel_InPlaceTranspose (int rowlen, hipDoubleComplex * transpose_matrix) {

    int col = blockIdx.x * (rowlen / gridDim.x) + threadIdx.x;
    int i;
    hipDoubleComplex temp;

    int row_index;
    int col_index;

    for (i = col; i < rowlen; i++)
    {
        row_index = (col * rowlen) + i;
        col_index = (i * rowlen) + col;
        temp = transpose_matrix[row_index];
        transpose_matrix[row_index] = transpose_matrix[col_index];
        transpose_matrix[col_index] = temp;
    }
}


typedef hipDoubleComplex data_t;

/* Create abstract data type for matrix */
typedef struct {
  long int len;
  data_t *data;
} matrix_rec, *matrix_ptr;


/*****************************************************************************/
int main(int argc, char *argv[])
{
  int OPTION;
  double interval(struct timespec start, struct timespec end);
  struct timespec time1, time2;

  int clock_gettime(clockid_t clk_id, struct timespec *tp);
  matrix_ptr new_matrix(long int len);
  int set_matrix_row_length(matrix_ptr m, long int row_len);
  long int get_matrix_length(matrix_ptr m);
  int init_matrix(matrix_ptr m, long int len);
  int zero_matrix(matrix_ptr m, long int len);
  void mmm_kij(matrix_ptr a, matrix_ptr b, matrix_ptr c);
  double fRand(double fMin, double fMax);
  int copy_matrix(data_t *original, data_t *copy, long int MAXSIZE);
  data_t *get_matrix_start(matrix_ptr m);
  
  long int i, j, k;
  long int time_sec, time_ns;
  float t;

  // GPU Timing variables
  hipEvent_t start, stop, start2, stop2;
  float elapsed_gpu, elapsed_just_FFT;

  // Arrays on GPU global memory
  data_t *FFT_gpu;
  data_t *FFT_host;



  /* declare and initialize the matrix structure */
  matrix_ptr fft_matrix = new_matrix(SIZE);
  fft_matrix->len = SIZE;
  generate_sin_points_c(fft_matrix->data, SIZE*SIZE, SAMPLING_RATE, FREQUENCY);

/*    printf("\n\n\nResult of GPU code\n");  
    for(i = 0; i < SIZE; ++i){
        for (j = 0; j < SIZE; ++j){
        printf("%.2lf j%.2lf   ", cuCreal(fft_matrix->data[i*SIZE+j]), cuCimag(fft_matrix->data[i*SIZE+j]) );
        }
        printf("\n");
    }
*/

  hipDoubleComplex *exptable = (hipDoubleComplex *) malloc((SIZE / 2) * sizeof(hipDoubleComplex));
  hipDoubleComplex *exptable_gpu;

  bool inverse = 0;
  hipDoubleComplex num;
  
  //printf("\n\n\n\n");
  for (i = 0; i < SIZE / 2; i++)
  {
    num = make_hipDoubleComplex( 0, (inverse ? 2: -2) * M_PI * i / SIZE );
    t = expf(num.x);
    sincos (num.y, &exptable[i].y, &exptable[i].x);
    exptable[i].x *= t;
    exptable[i].y *= t;
    //printf("%.2lf j%.2lf     ", cuCreal(exptable[i]), cuCimag(exptable[i]) );
  }
  //printf("\n\n\n\n\n");

  // Allocate CPU memory
  size_t allocSize = (SIZE*SIZE) * sizeof(hipDoubleComplex);
  size_t allocSize2 = (SIZE/2) * sizeof(hipDoubleComplex);

  FFT_host       = (hipDoubleComplex *) malloc(allocSize);
  FFT_gpu        = (hipDoubleComplex *) malloc(allocSize);

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));


  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&start2);
  hipEventCreate(&stop2);

  // Record event on the default stream
  hipEventRecord(start, 0);

  // Allocate GPU memory

  CUDA_SAFE_CALL(hipMalloc((void **)&FFT_gpu, allocSize));
  CUDA_SAFE_CALL(hipMalloc( (void **)&exptable_gpu, allocSize2 ));
  CUDA_SAFE_CALL(hipMemcpy(FFT_gpu,fft_matrix->data,allocSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(exptable_gpu,exptable,allocSize2,hipMemcpyHostToDevice));
  
  // Launch the kernels to make 2D FFT happen!

  dim3 dimGrid(NUM_BLOCKS,1,1);
  dim3 dimBlock(SIZE/NUM_BLOCKS,1,1);

  hipEventRecord(start2, 0);
  
  kernel_FFT <<<dimGrid, dimBlock>>>(SIZE, exptable_gpu, FFT_gpu);
  kernel_InPlaceTranspose <<<dimGrid, dimBlock>>>(SIZE, FFT_gpu);
  kernel_FFT <<<dimGrid, dimBlock>>>(SIZE, exptable_gpu, FFT_gpu);
  kernel_InPlaceTranspose <<<dimGrid, dimBlock>>>(SIZE, FFT_gpu);

  hipEventRecord(stop2, 0);
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&elapsed_just_FFT, start2, stop2);

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());
  CUDA_SAFE_CALL(hipMemcpy(FFT_host,FFT_gpu,allocSize,hipMemcpyDeviceToHost));


  CUDA_SAFE_CALL(hipFree(FFT_gpu));
  
  // Stop and destroy the timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  printf("\nGPU total time: %f (msec), GPU MMM time: %f (msec)\n", elapsed_gpu, elapsed_just_FFT);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipEventDestroy(start2);
  hipEventDestroy(stop2);


/*  printf("\n\n\nResult of GPU code\n");  
    for(i = 0; i < SIZE; ++i){
        for (j = 0; j < SIZE; ++j){
        printf("%.2lf j%.2lf   ", cuCreal(FFT_host[i*SIZE+j]), cuCimag(FFT_host[i*SIZE+j]) );
        }
        printf("\n");
    }
*/

}/* end main */













/**********************************************/

/* Returns a random number between fMin and fMax */
double fRand(double fMin, double fMax)
{
  double f = (double)random() / RAND_MAX;
  return fMin + f * (fMax - fMin);
}


/* Create matrix of specified length */
matrix_ptr new_matrix(long int len)
{
  long int i;

  /* Allocate and declare header structure */
  matrix_ptr result = (matrix_ptr) malloc(sizeof(matrix_rec));
  if (!result) return NULL;  /* Couldn't allocate storage */
  result->len = len;

  /* Allocate and declare array */
  if (len > 0) {
    data_t *data = (data_t *) calloc(len*len, sizeof(hipDoubleComplex));
    if (!data) {
	  free((void *) result);
	  printf("\n COULDN'T ALLOCATE %ld BYTES STORAGE \n", result->len);
	  return NULL;  /* Couldn't allocate storage */
	}
	result->data = data;
  }
  else result->data = NULL;

  return result;
}

/* Set length of matrix */
int set_matrix_row_length(matrix_ptr m, long int row_len)
{
  m->len = row_len;
  return 1;
}

/* Return length of matrix */
long int get_matrix_length(matrix_ptr m)
{
  return m->len;
}

/* initialize matrix */
int init_matrix(matrix_ptr m, long int len)
{
  long int i;

  if (len > 0) {
    m->len = len;
    for (i = 0; i < len*len; i++)
      m->data[i] = make_hipDoubleComplex( fRand((double)(5.0),(double)(15.0)) , 0);
    return 1;
  }
  else return 0;
}

/* initialize matrix */
int zero_matrix(matrix_ptr m, long int len)
{
  long int i,j;

  if (len > 0) {
    m->len = len;
    for (i = 0; i < len*len; i++)
      m->data[i] = make_hipDoubleComplex( IDENT, 0 );
    return 1;
  }
  else return 0;
}

int copy_matrix(data_t *original, data_t *copy, long int MAXSIZE)
{
  int i;

  printf("Made it here!\n");
  for (i = 0; i < MAXSIZE * MAXSIZE; i++)
  {
    printf("%d, ", i);
    copy[i] = original[i];
  }
  printf("\n");
  return 1;
}

data_t *get_matrix_start(matrix_ptr m)
{
  return m->data;
}

/*************************************************/

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}